#include "hip/hip_runtime.h"
/*
solution:	raytracing
project:	rtengine2
file:		kernel.cpp
author:		cj
*/

#include "../cmath/ray.cuh"
#include "../cmath/sphere.cuh"
#include "../cmath/plane.cuh"
#include "../cmath/intersections.cuh"
#include "hip/hip_runtime.h"
#include "rendertarget.h"
#include "world.h"
#include "kernel.h"

namespace RT2 {

	struct WInfo {
		CM::Ray ray;
		CM::IS::Info minfo;
		CM::Color color;
		float refIndex;
		float reflectivity;
		float refractivity;
		float transparency;
	};

	__device__ bool WorldIntersects(CM::Ray ray, WInfo* info, Sphere* spheres, unsigned numSpheres) {
		bool hit = false;
		float minDist = 0.0f;
		int minIndex = 0;
		int i = 0;

		const float eps = 0.001f;

		CM::IS::Info minfo;
		for(; i < numSpheres; ++i) {
			Sphere sphere = spheres[i];
			if(CM::IS::IntersectAnalytically(ray, sphere.geometry, &minfo)) {
				float dist = minfo.distance;
				if(abs(dist) > eps) {
					if(!hit) {
						hit = true;
						minDist = dist;
						minIndex = i;
						info->minfo = minfo;
					} else if(minDist > dist) {
						minDist = dist;
						minIndex = i;
						info->minfo = minfo;
					}
				}
			}
		}

		if(hit) {
			info->color = spheres[minIndex].color;
			info->ray = ray;
			info->reflectivity = spheres[minIndex].reflectivity;
			info->refractivity = spheres[minIndex].refractivity;
			info->transparency = spheres[minIndex].transparency;
			info->refIndex = spheres[minIndex].refIndex;
		}

		// hard coded floor intersection
		const CM::Plane floorPlane(CM::Vector3(0.0f, 0.0f, 1.0f), -0.5f);
		bool planeHit = false;
		if(CM::IS::IntersectAnalytically(ray, floorPlane, &minfo)) {
			float dist = minfo.distance;
			if(abs(dist) > eps) {
				if(!hit) {
					hit = true;
					minDist = dist;
					info->minfo = minfo;
					planeHit = true;
				} else if(minDist > dist) {
					minDist = dist;
					info->minfo = minfo;
					planeHit = true;
				}
			}
		}

		if(planeHit) {
			info->color = CM::Color(1.0f, 1.0f, 1.0f);
			info->ray = ray;
			info->reflectivity = 0.2f;
			info->refractivity = 0.0f;
			info->transparency = 0.0f;
		}

		return hit;
	}

	__device__ CM::Color Miss(void) {
		return CM::Color(0.8f, 0.7f, 1.0f);
	}

	template<unsigned DEPTH>
	__device__ CM::Color Hit(WInfo* info,
								Sphere* spheres, unsigned numSpheres,
								Globals* globals) 
	{
		CM::Vector3 where = info->minfo.where;
		CM::Vector3 normal = info->minfo.normal;

		CM::Vector3 dir = CM::Normalize(globals->lightPos - where);
		CM::Ray shadowRay(where, dir);

		CM::Color fragColor = CM::Color(0.0f, 0.0f, 0.0f);

		// cast shadow ray
		WInfo linfo;
		if(!WorldIntersects(shadowRay, &linfo, spheres, numSpheres)) {
			// diffuse
			float d = CM::Clamp(CM::Dot(dir, normal), 0.0f, 1.0f);
			CM::Color diff = info->color * d;
			fragColor += diff;

			// specular
			const float shininess = 20.0f;
			CM::Color spec = CM::Color(0.0f, 0.0f, 0.0f);
			if(0.0f < d) {
				CM::Vector3 s = CM::Normalize(dir + globals->eyePos - where);
				d = CM::Max(CM::Dot(s, normal), 0.0f);
				spec = powf(d, shininess) * CM::Color(1.0f, 1.0f, 1.0f);
			}
			fragColor += spec;
		}

		// cast reflection ray
		const float reflectivity = info->reflectivity;
		if(0.0f < reflectivity) {
			CM::Ray reflectionRay(where, CM::ReflectNormalized(info->ray.direction, normal));
			if(WorldIntersects(reflectionRay, &linfo, spheres, numSpheres)) {
				fragColor += reflectivity * Hit<DEPTH - 1>(&linfo, spheres, numSpheres, globals);
			} else {
				fragColor += reflectivity * Miss();
			}
		}

		// cast refraction ray
		/*const float refractivity = info->refractivity;
		if(0.0f < refractivity) {
			CM::Ray refractionRay(where, CM::RefractNormalized(info->ray.direction, normal, 1.003f, info->refIndex));
			if(WorldIntersects(refractionRay, &linfo, spheres, numSpheres)) {
				fragColor += refractivity * Hit<DEPTH - 1>(&linfo, spheres, numSpheres, globals);
			} else {
				fragColor += refractivity * Miss();
			}
		}*/

		// cast transparency ray
		/*
		const float transparency = info->transparency;
		if(0.0f < transparency) {
			CM::Ray transparencyRay(where, info->ray.direction);
			if(WorldIntersects(transparencyRay, &linfo, spheres, numSpheres)) {
				fragColor += transparency * Hit<DEPTH - 1>(&linfo, spheres, numSpheres, globals);
			} else {
				fragColor += transparency * Miss();
			}
		}
		*/

		return fragColor;
	}

	template<>
	__device__ CM::Color Hit<0>(WInfo*, Sphere*, unsigned, Globals*) {
		return CM::Color(0.0f, 0.0f, 0.0f);
	}

	__device__ CM::Color Saturate(CM::Color color) {
		if(1.0f <= color.r) color.r = 1.0f;
		if(1.0f <= color.g) color.g = 1.0f;
		if(1.0f <= color.b) color.b = 1.0f;
		if(1.0f <= color.a) color.a = 1.0f;
		return color;
	}

	__global__ void Kernel(COM::byte_t* renderTarget, unsigned pitch, float dx, float dy,
							Sphere* spheres, unsigned numSpheres,
							Globals* globals) 
	{
		int x = blockIdx.x * 16 + threadIdx.x;
		int y = blockIdx.y * 16 + threadIdx.y;
		CM::Color* pixel = (CM::Color*)(renderTarget + y * pitch) + x;

		float fx = -1.0f + x * dx;
		float fy = -1.0f + y * dy;
		const float viewDistance = 10.0f;

		CM::Vector3 viewDir = CM::Normalize(CM::Transform(globals->invWorld, CM::Vector3(fx, fy, -viewDistance)));
		CM::Vector3 eyePos = CM::Transform(globals->invWorld, CM::Vector3(0.0f, 0.0f, viewDistance));

		globals->lightPos = CM::Transform(globals->invWorld, globals->lightPos);

		CM::Ray ray;
		ray.origin = eyePos;
		ray.direction = viewDir;

		globals->eyePos = eyePos;

		CM::Color fragColor;

		WInfo info;
		if(WorldIntersects(ray, &info, spheres, numSpheres)) {
			fragColor = Hit<2>(&info, spheres, numSpheres, globals);
		} else {
			fragColor = Miss();
		}

		*pixel = Saturate(fragColor);
	}

	void RunKernel(RenderTarget& renderTarget, RT2::World& world) {
		const float dx = 2.0f / renderTarget.Width();
		const float dy = 2.0f / renderTarget.Height();

		glBindTexture(GL_TEXTURE_2D, 0);
		renderTarget.Map();
		Kernel<<<dim3(64, 64), dim3(16, 16)>>>(renderTarget.GetDeviceMemory(), renderTarget.GetPitch(), dx, dy,
												world.GetSphereMemory(), world.GetSize(),
												world.GetGlobalsMemory());
		renderTarget.Unmap();

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			COM::log << "running kernel failed: "
				<< hipGetErrorString(error) << std::endl;
		}
	}

} // namespace RT2